#include "mex.h"
#include "tcbf.h"  // include the header

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
  if (nrhs != 7) {
    mexErrMsgIdAndTxt("beamform:InvalidInput",
                      "Seven inputs required: a_matrix, rf, bf, pixels, frames, samples, device_id");
  }

  std::string path_a_matrix = mxArrayToString(prhs[0]);
  std::string path_rf = mxArrayToString(prhs[1]);
  std::string path_bf = mxArrayToString(prhs[2]);
  size_t pixels = mxGetScalar(prhs[3]);
  size_t frames = mxGetScalar(prhs[4]);
  size_t samples = mxGetScalar(prhs[5]);
  unsigned device_id = mxGetScalar(prhs[6]);

  cu::init();
  cu::Device device(device_id);
  cu::Context context(hipDeviceScheduleBlockingSync, device);
  cu::Stream stream;

  tcbf::Beamformer beamformer(pixels, frames, samples, device, stream);
  cu::HostMemory RF(2 * frames * samples);
  cu::HostMemory BF(2 * pixels * frames * sizeof(unsigned));

  beamformer.read_A_matrix(path_a_matrix);
  beamformer.read_RF(RF, path_rf);
  beamformer.process(RF, BF);
  beamformer.write_BF(BF, path_bf);

  mwSize dims[2] = {2 * pixels, frames};
  mxArray *outArray = mxCreateNumericArray(2, dims, mxINT32_CLASS, mxREAL);
  int32_t *outData = static_cast<int32_t *>(mxGetData(outArray));

  // Copy the data
  std::memcpy(outData, BF, totalElements * sizeof(int32_t));

  // Assign output
  plhs[0] = outArray;

  int status = 0;

  if (nlhs > 1) {
    plhs[1] = mxCreateDoubleScalar((double)status);
  }
}

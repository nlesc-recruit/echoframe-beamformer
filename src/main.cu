#include <fstream>
#include <iostream>

#include "tcbf.h"

int main() {
  cu::init();
  cu::Device device(0);
  cu::Context context(hipDeviceScheduleBlockingSync, device);
  cu::Stream stream;

  const size_t pixels = 38880;
  const size_t frames = 8041;
  const size_t samples = 524288;

  tcbf::Beamformer beamformer(pixels, frames, samples, device, stream);
  cu::HostMemory RF(2 * frames * samples);
  cu::HostMemory BF(2 * pixels * frames * sizeof(unsigned));

  beamformer.read_A_matrix("/var/scratch/oostrum/cube_data/gemm/sign_demo/"
                           "A_packed_transposed_conjugated_64_256.bin");
  beamformer.read_RF(
      RF,
      "/var/scratch/oostrum/cube_data/gemm/sign_demo/RF_full_524288_8041.bin");

  beamformer.process(RF, BF);

  std::ofstream out("/var/scratch/oostrum/cube_data/gemm/sign_demo/BF.bin",
                    std::ios::binary | std::ios::out);
  if (!out) {
    throw std::runtime_error("Failed to open output file");
  }

  // real part
  for (size_t frame = 0; frame < frames_data; frame++) {
    out.write(static_cast<char *>(BF) + frame * pixels * sizeof(int),
              pixels_data * sizeof(int));
  }
  // imag part
  for (size_t frame = frames; frame < frames + frames_data; frame++) {
    out.write(static_cast<char *>(BF) + frame * pixels * sizeof(int),
              pixels_data * sizeof(int));
  }
}
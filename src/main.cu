#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <limits.h>

#include <ccglib/ccglib.hpp>
#include <cudawrappers/cu.hpp>

template <typename T> inline T align(const T a, const T b) {
  return b * (a / b + ((a % b) != 0));
}

template <typename T>
void read_file(const std::string path, char *data, const size_t M,
               const size_t N, const size_t M_padded, const size_t N_padded) {
  std::ifstream in(path, std::ios::binary | std::ios::in);
  if (!in) {
    throw std::runtime_error("Failed to open input file: " + path);
  }

  for (size_t m = 0; m < M; m++) {
    const size_t byte_offset = m * N_padded * sizeof(T);

    in.read(data + byte_offset, N * sizeof(T));
  }
}

void read_file(const std::string path, cu::HostMemory &data,
               const size_t bytes) {
  std::ifstream in(path, std::ios::binary | std::ios::in);
  if (!in) {
    throw std::runtime_error("Failed to open input file: " + path);
  }

  in.read(static_cast<char *>(data), bytes);
}

template <typename T>
void write_file(const std::string path, char *data, const size_t M,
                const size_t N, const size_t M_padded, const size_t N_padded) {
  std::ofstream out(path, std::ios::binary | std::ios::out);
  if (!out) {
    throw std::runtime_error("Failed to open output file: " + path);
  }

  // real part
  for (size_t m = 0; m < M; m++) {
    const size_t byte_offset = m * N_padded * sizeof(T);
    out.write(data + byte_offset, N * sizeof(T));
  }
  // imag part
  for (size_t m = M_padded; m < M_padded + M; m++) {
    const size_t byte_offset = m * N_padded * sizeof(T);
    out.write(data + byte_offset, N * sizeof(T));
  }
}

int main() {
  cu::init();
  cu::Device device(0);
  cu::Context context(hipDeviceScheduleBlockingSync, device);
  cu::Stream stream;

  const std::string path = "/var/scratch/oostrum/cube_data/gemm/sign_demo/";

  const unsigned complex = 2;
  const size_t beams_data = 38880;    // M axis
  const size_t frames_data = 8041;    // N axis
  const size_t samples_data = 524288; // K axis
  const size_t nr_bits = 1;

  // initialize all ccglib objects -- this triggers compilation as well
  const auto gemm_precision = ccglib::mma::int1;
  const auto gemm_variant = ccglib::mma::opt;
  // obtain matrix tile sizes from GEMM, order is M, N, K
  const dim3 dimensions =
      ccglib::mma::GEMM::GetDimensions(gemm_precision, gemm_variant);

  // for ccglib only supports matrices that are multiples of the GEMM
  // dimensions, take care of padding in this code for now const size_t beams =
  // align(beams_data, static_cast<size_t>(dimensions.x)); const size_t frames =
  // align(frames_data,  static_cast<size_t>(dimensions.y)); const size_t
  // samples = align(samples_data,  static_cast<size_t>(dimensions.z));
  constexpr size_t beams = 38912;
  constexpr size_t frames = 8064;
  constexpr size_t samples = 524288;
  std::cout << beams_data << " " << frames_data << " " << samples_data
            << std::endl;
  std::cout << beams << " " << frames << " " << samples << std::endl;

  const size_t bytes_a_matrix = beams * samples * complex;
  const size_t bytes_rf = frames * samples * complex;
  const size_t bytes_a_matrix_packed = bytes_a_matrix / CHAR_BIT;
  const size_t bytes_rf_packed = bytes_rf / CHAR_BIT;

  ccglib::packing::Packing pack_rf(frames * samples, device, stream);
  ccglib::transpose::Transpose transpose_rf(
      1, frames, samples, dimensions.y, dimensions.z, nr_bits, device, stream);
  ccglib::mma::GEMM gemm(1, beams, frames, samples, nr_bits, device, stream,
                         gemm_precision, gemm_variant);

  // the inputs are sign data stored as one byte per sample
  // read prepared A matrix
  cu::HostMemory A(bytes_a_matrix_packed);
  read_file(path + "A_packed_transposed_64_256.bin", A, bytes_a_matrix_packed);
  cu::DeviceMemory d_A_matrix_trans(bytes_a_matrix_packed);
  stream.memcpyHtoDAsync(d_A_matrix_trans, A, bytes_a_matrix_packed);

  auto RF_real = new char[frames][samples];
  auto RF_imag = new char[frames][samples];

  std::cout << "Loading RF - real" << std::endl;
  read_file<char>(path + "RF_Real_524288_8041.bin", &RF_real[0][0], frames_data,
                  samples_data, frames, samples);
  std::cout << "Loading RF - imag" << std::endl;
  read_file<char>(path + "RF_Imag_524288_8041.bin", &RF_imag[0][0], frames_data,
                  samples_data, frames, samples);
  std::cout << "Data loaded" << std::endl;

  // step 1. packing
  std::cout << "Packing RF" << std::endl;
  cu::DeviceMemory d_RF(bytes_rf_packed);
  // get a pointer to start of RF imag
  hipDeviceptr_t RF_offset = reinterpret_cast<hipDeviceptr_t>(
      reinterpret_cast<char *>(static_cast<hipDeviceptr_t>(d_RF)) +
      bytes_rf_packed / complex);
  cu::DeviceMemory d_RF_imag_only(RF_offset, bytes_rf_packed / complex);

  cu::HostMemory RF_imag_host(RF_imag, bytes_rf / complex);
  cu::HostMemory RF_real_host(RF_real, bytes_rf / complex);
  pack_rf.Run(RF_real_host, d_RF, ccglib::packing::pack);
  pack_rf.Run(RF_imag_host, d_RF_imag_only, ccglib::packing::pack);

  // step 2. transpose
  std::cout << "Transpose RF" << std::endl;
  cu::DeviceMemory d_RF_trans(d_RF.size());
  transpose_rf.Run(d_RF, d_RF_trans);

  // step 3. beamform
  std::cout << "GEMM" << std::endl;
  cu::DeviceMemory d_BF(beams * frames * complex * sizeof(int));
  gemm.Run(d_A_matrix_trans, d_RF_trans, d_BF);

  cu::HostMemory BF(beams * frames * complex * sizeof(int));
  stream.memcpyDtoHAsync(BF, d_BF, beams * frames * complex * sizeof(int));
  stream.synchronize();

  const std::string output_file = path + "BF.bin";
  std::cout << "Writing output to " << output_file << std::endl;
  // row major
  // write_file<int>(output_file, BF, beams_data, frames_data, beams, frames);
  // col major
  write_file<int>(output_file, static_cast<char *>(BF), frames_data, beams_data,
                  frames, beams);
}
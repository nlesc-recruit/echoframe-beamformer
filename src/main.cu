#include <cxxopts.hpp>
#include <fstream>
#include <iostream>

#include "tcbf.h"

cxxopts::Options create_commandline_parser(const char *argv[]) {
  cxxopts::Options options(argv[0], "Echoframe standalone beamformer");

  options.add_options()("a_matrix", "Full path to input prepared A matrix",
                        cxxopts::value<std::string>())(
      "rf", "Full path to input RF", cxxopts::value<std::string>())(
      "bf", "Full path to output BF", cxxopts::value<std::string>())(
      "pixels", "Number of pixels", cxxopts::value<size_t>())(
      "frames", "Number of frames", cxxopts::value<size_t>())(
      "samples", "Number of samples", cxxopts::value<size_t>())(
      "device", "GPU device ID",
      cxxopts::value<unsigned>()->default_value(std::to_string(0)))(
      "h,help", "Print help");

  return options;
}

cxxopts::ParseResult parse_commandline(int argc, const char *argv[]) {
  cxxopts::Options options = create_commandline_parser(argv);

  try {
    cxxopts::ParseResult result = options.parse(argc, argv);

    if (result.count("help")) {
      std::cout << options.help() << std::endl;
      exit(EXIT_SUCCESS);
    }

    std::vector<std::string> required_options{"a_matrix", "rf",     "bf",
                                              "pixels",   "frames", "samples"};
    for (auto &opt : required_options) {
      if (!result.count(opt)) {
        std::cerr << "Required argument missing: " << opt << std::endl;
        std::cerr << "Run " << argv[0] << " -h for help" << std::endl;
        exit(EXIT_FAILURE);
      }
    }

    return result;
  } catch (const cxxopts::exceptions::exception &err) {
    std::cerr << "Error parsing commandline: " << err.what() << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main(int argc, const char *argv[]) {
  cxxopts::ParseResult cmdline = parse_commandline(argc, argv);
  const std::string path_a_matrix = cmdline["a_matrix"].as<std::string>();
  const std::string path_rf = cmdline["rf"].as<std::string>();
  const std::string path_bf = cmdline["bf"].as<std::string>();
  const size_t pixels = cmdline["pixels"].as<size_t>();
  const size_t frames = cmdline["frames"].as<size_t>();
  const size_t samples = cmdline["samples"].as<size_t>();
  const unsigned device_id = cmdline["device"].as<unsigned>();

  cu::init();
  cu::Device device(device_id);
  cu::Context context(hipDeviceScheduleBlockingSync, device);
  cu::Stream stream;

  tcbf::Beamformer beamformer(pixels, frames, samples, device, stream);
  cu::HostMemory RF(2 * frames * samples);
  cu::HostMemory BF(2 * pixels * frames * sizeof(unsigned));

  beamformer.read_A_matrix(path_a_matrix);
  beamformer.read_RF(RF, path_rf);
  beamformer.process(RF, BF);
  beamformer.write_BF(BF, path_bf);
}
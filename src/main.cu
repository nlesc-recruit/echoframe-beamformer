#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include <ccglib/helper.h>
#include <tcbf.h>

inline size_t align(size_t a, size_t b) {
  return b * ccglib::helper::ceildiv(a, b);
}

int main() {
  cu::init();
  cu::Device device(0);
  cu::Context context(hipDeviceScheduleBlockingSync, device);
  cu::Stream stream;

  const size_t pixels_data = 38880;
  const size_t frames_data = 8041;
  const size_t samples_data = 524288;

  dim3 tile_sizes =
      ccglib::mma::GEMM::GetDimensions(ccglib::mma::int1, ccglib::mma::opt);

  const size_t pixels = align(pixels_data, tile_sizes.x);
  const size_t frames = align(frames_data, tile_sizes.y);
  const size_t samples = align(samples_data, tile_sizes.z);

  tcbf::Beamformer beamformer(pixels, frames, samples, device, stream);
  cu::HostMemory RF(beamformer.bytesRF_);
  cu::HostMemory BF(beamformer.bytesBF_);

  beamformer.read_A_matrix("/var/scratch/oostrum/cube_data/gemm/sign_demo/"
                           "A_packed_transposed_conjugated_64_256.bin");
  beamformer.read_RF(
      RF,
      "/var/scratch/oostrum/cube_data/gemm/sign_demo/RF_full_524288_8041.bin",
      frames_data, samples_data);
  beamformer.process(RF, BF);

  std::ofstream out("/var/scratch/oostrum/cube_data/gemm/sign_demo/BF.bin",
                    std::ios::binary | std::ios::out);
  if (!out) {
    throw std::runtime_error("Failed to open output file");
  }

  // real part
  for (size_t frame = 0; frame < frames_data; frame++) {
    out.write(static_cast<char *>(BF) + frame * pixels * sizeof(int),
              pixels_data * sizeof(int));
  }
  // imag part
  for (size_t frame = frames; frame < frames + frames_data; frame++) {
    out.write(static_cast<char *>(BF) + frame * pixels * sizeof(int),
              pixels_data * sizeof(int));
  }
}
#include "hip/hip_runtime.h"
#include <ccglib/helper.h>
#include <limits.h>
#include <tcbf.h>

#include <ccglib/ccglib.hpp>
#include <cxxopts.hpp>
#include <fstream>
#include <iostream>

inline size_t align(size_t a, size_t b) { return b * ccglib::helper::ceildiv(a, b); }

cxxopts::Options create_commandline_parser(const char *argv[]) {
  cxxopts::Options options(argv[0], "Echoframe beamformer A matrix preparation");

  options.add_options()("a_matrix_in", "Full path to input A matrix", cxxopts::value<std::string>())(
      "a_matrix_out", "Full path to output A matrix", cxxopts::value<std::string>())(
      "pixels", "Number of pixels", cxxopts::value<size_t>())("samples", "Number of samples", cxxopts::value<size_t>())(
      "device", "GPU device ID", cxxopts::value<unsigned>()->default_value(std::to_string(0)))("h,help", "Print help");

  return options;
}

cxxopts::ParseResult parse_commandline(int argc, const char *argv[]) {
  cxxopts::Options options = create_commandline_parser(argv);

  try {
    cxxopts::ParseResult result = options.parse(argc, argv);

    if (result.count("help")) {
      std::cout << options.help() << std::endl;
      exit(EXIT_SUCCESS);
    }

    std::vector<std::string> required_options{"a_matrix_in", "a_matrix_out", "pixels", "samples"};
    for (auto &opt : required_options) {
      if (!result.count(opt)) {
        std::cerr << "Required argument missing: " << opt << std::endl;
        std::cerr << "Run " << argv[0] << " -h for help" << std::endl;
        exit(EXIT_FAILURE);
      }
    }

    return result;
  } catch (const cxxopts::exceptions::exception &err) {
    std::cerr << "Error parsing commandline: " << err.what() << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main(int argc, const char *argv[]) {
  cxxopts::ParseResult cmdline = parse_commandline(argc, argv);
  const std::string path_a_matrix_in = cmdline["a_matrix_in"].as<std::string>();
  const std::string path_a_matrix_out = cmdline["a_matrix_out"].as<std::string>();
  const size_t pixels = cmdline["pixels"].as<size_t>();
  const size_t samples = cmdline["samples"].as<size_t>();
  const unsigned device_id = cmdline["device"].as<unsigned>();
  const size_t complex = 2;

  cu::init();
  cu::Device device(device_id);
  cu::Context context(hipDeviceScheduleBlockingSync, device);
  cu::Stream stream;

  // tile size in beams, frames, samples axes
  dim3 tile_sizes = ccglib::mma::GEMM::GetDimensions(ccglib::mma::int1, ccglib::mma::opt);

  const size_t pixels_padded = align(pixels, tile_sizes.x);
  const size_t samples_padded = align(samples, tile_sizes.z);

  // factor 2 for complex
  const size_t bytes_a_matrix = complex * pixels_padded * samples_padded;
  const size_t bytes_a_matrix_packed = bytes_a_matrix / CHAR_BIT;

  // Read data from disk
  // row-by-row to handle padding
  cu::HostMemory a_matrix_host(bytes_a_matrix);
  std::ifstream in(path_a_matrix_in, std::ios::binary | std::ios::in);
  if (!in) {
    throw std::runtime_error("Failed to open input file: " + path_a_matrix_in);
  }
  for (size_t c = 0; c < complex; c++) {
    for (size_t pixel = 0; pixel < pixels; pixel++) {
      in.read(static_cast<char *>(a_matrix_host) + c * pixels_padded * samples_padded + pixel * samples_padded,
              samples);
    }
  }
  in.close();

  // conjugate
  std::cout << "Conjugate" << std::endl;
#pragma omp parallel for collapse(2)
  for (size_t pixel = 0; pixel < pixels; pixel++) {
    for (size_t sample = 0; sample < samples; sample++) {
      const size_t idx = pixels_padded * samples_padded + pixel * samples_padded + sample;
      static_cast<char *>(a_matrix_host)[idx] = 1 - static_cast<char *>(a_matrix_host)[idx];
    }
  }

  // Device memory for output packed data
  cu::DeviceMemory d_a_matrix_packed(bytes_a_matrix_packed);
  d_a_matrix_packed.zero(bytes_a_matrix_packed);
  // Device memory for transposed data
  cu::DeviceMemory d_a_transposed(bytes_a_matrix_packed);

  // chunk of input data on device in case it doesn't fit in GPU memory
  // get available GPU memory (after allocating other device memory)
  // use at most 80% of available memory
  size_t bytes_per_chunk = .8 * context.getFreeMemory();
  // packing kernel uses at most 1024 threads per block (and should be a power of 2), each thread processes one byte
  // round to multiple of a kilobyte such that it correspond to a whole number of blocks
  bytes_per_chunk = 1024 * (bytes_per_chunk / 1024);
  if (bytes_per_chunk > bytes_a_matrix) {
    bytes_per_chunk = bytes_a_matrix;
  }
  cu::DeviceMemory d_a_chunk(bytes_per_chunk);
  d_a_chunk.zero(bytes_per_chunk);

  // process, complex-first for now
  std::cout << "Packing" << std::endl;
  for (size_t byte_start = 0; byte_start < bytes_a_matrix; byte_start += bytes_per_chunk) {
    size_t local_nbytes = bytes_per_chunk;
    // correct nbytes in last chunk
    if (byte_start + local_nbytes > bytes_a_matrix) {
      local_nbytes = bytes_a_matrix - byte_start;
      // ensure any padded region is set to zero
      d_a_chunk.zero(bytes_per_chunk);
    }
    // copy chunk to device
    stream.memcpyHtoDAsync(d_a_chunk, static_cast<char *>(a_matrix_host) + byte_start, local_nbytes);
    // get device memory slice for this chunk in a_packed
    cu::DeviceMemory d_a_packed_chunk(d_a_matrix_packed, byte_start / CHAR_BIT, local_nbytes / CHAR_BIT);
    // run packing kernel
    ccglib::packing::Packing packing(local_nbytes, device, stream);
    packing.Run(d_a_chunk, d_a_packed_chunk, ccglib::packing::pack, ccglib::packing::complex_first);
  }

  // transpose
  std::cout << "Transpose" << std::endl;
  ccglib::transpose::Transpose transpose(1, pixels_padded, samples_padded, tile_sizes.x, tile_sizes.z, 1, device,
                                         stream);
  transpose.Run(d_a_matrix_packed, d_a_transposed);

  // copy output to host
  std::cout << "Copy to host" << std::endl;
  cu::HostMemory a_matrix_output(bytes_a_matrix_packed);
  stream.memcpyDtoHAsync(a_matrix_output, d_a_transposed, bytes_a_matrix_packed);
  stream.synchronize();

  // write to disk
  std::cout << "Write to disk" << std::endl;
  std::ofstream out(path_a_matrix_out, std::ios::binary | std::ios::out);
  if (!out) {
    throw std::runtime_error("Failed to open output file: " + path_a_matrix_out);
  }
  out.write(static_cast<char *>(a_matrix_output), bytes_a_matrix_packed);
}

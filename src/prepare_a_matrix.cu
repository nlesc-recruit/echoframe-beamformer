#include "hip/hip_runtime.h"
#include <ccglib/helper.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <tcbf.h>

#include <ccglib/ccglib.hpp>
#include <cxxopts.hpp>
#include <fstream>
#include <iostream>

__global__ void conjugate_1bit(unsigned *data, const size_t n_bytes) {
  size_t tid = threadIdx.x + blockDim.x * static_cast<size_t>(blockIdx.x);
  const size_t n_elements = n_bytes / sizeof(unsigned);
  if (tid >= n_elements) {
    return;
  }
  data[tid] = ~data[tid];
}

inline size_t align(size_t a, size_t b) { return b * ccglib::helper::ceildiv(a, b); }

cxxopts::Options create_commandline_parser(const char *argv[]) {
  cxxopts::Options options(argv[0], "Echoframe beamformer A matrix preparation");

  options.add_options()("a_matrix_in", "Full path to input A matrix", cxxopts::value<std::string>())(
      "a_matrix_out", "Full path to output A matrix", cxxopts::value<std::string>())(
      "pixels", "Number of pixels", cxxopts::value<size_t>())("samples", "Number of samples", cxxopts::value<size_t>())(
      "device", "GPU device ID", cxxopts::value<unsigned>()->default_value(std::to_string(0)))("h,help", "Print help");

  return options;
}

cxxopts::ParseResult parse_commandline(int argc, const char *argv[]) {
  cxxopts::Options options = create_commandline_parser(argv);

  try {
    cxxopts::ParseResult result = options.parse(argc, argv);

    if (result.count("help")) {
      std::cout << options.help() << std::endl;
      exit(EXIT_SUCCESS);
    }

    std::vector<std::string> required_options{"a_matrix_in", "a_matrix_out", "pixels", "samples"};
    for (auto &opt : required_options) {
      if (!result.count(opt)) {
        std::cerr << "Required argument missing: " << opt << std::endl;
        std::cerr << "Run " << argv[0] << " -h for help" << std::endl;
        exit(EXIT_FAILURE);
      }
    }

    return result;
  } catch (const cxxopts::exceptions::exception &err) {
    std::cerr << "Error parsing commandline: " << err.what() << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main(int argc, const char *argv[]) {
  cxxopts::ParseResult cmdline = parse_commandline(argc, argv);
  const std::string path_a_matrix_in = cmdline["a_matrix_in"].as<std::string>();
  const std::string path_a_matrix_out = cmdline["a_matrix_out"].as<std::string>();
  const size_t pixels = cmdline["pixels"].as<size_t>();
  const size_t samples = cmdline["samples"].as<size_t>();
  const unsigned device_id = cmdline["device"].as<unsigned>();

  cu::init();
  cu::Device device(device_id);
  cu::Context context(hipDeviceScheduleBlockingSync, device);
  cu::Stream stream;

  // tile size in beams, frames, samples axes
  dim3 tile_sizes = ccglib::mma::GEMM::GetDimensions(ccglib::mma::int1, ccglib::mma::opt);

  const size_t pixels_padded = align(pixels, tile_sizes.x);
  const size_t samples_padded = align(samples, tile_sizes.z);

  // factor 2 for complex
  // host is unpadded, device is always padded
  const size_t bytes_a_matrix = 2UL * pixels * samples;
  const size_t bytes_a_matrix_packed = 2UL * pixels_padded * samples_padded / CHAR_BIT;

  // Read data from disk
  cu::HostMemory a_matrix_host(bytes_a_matrix);
  std::ifstream in(path_a_matrix_in, std::ios::binary | std::ios::in);
  if (!in) {
    throw std::runtime_error("Failed to open input file: " + path_a_matrix_in);
  }
  in.read(static_cast<char *>(a_matrix_host), bytes_a_matrix);
  in.close();

  // Device memory for output packed data
  cu::DeviceMemory d_a_matrix_packed(bytes_a_matrix_packed);
  d_a_matrix_packed.zero(bytes_a_matrix_packed);
  // Device memory for transposed data
  cu::DeviceMemory d_a_transposed(bytes_a_matrix_packed);

  // chunk of input data on device in case it doesn't fit in GPU memory
  // get available GPU memory (after allocating other device memory)
  // use at most 80% of available memory
  size_t chunk_size = .8 * context.getFreeMemory();
  size_t pixels_per_chunk = chunk_size / (samples_padded);
  if (pixels_per_chunk > pixels) {
    pixels_per_chunk = pixels;
  }
  chunk_size = pixels_per_chunk * samples_padded;
  cu::DeviceMemory d_a_chunk(chunk_size);
  d_a_chunk.zero(chunk_size);

  // process, complex-first for now
  // first real, then imag part
  std::cout << "Start of processing" << std::endl;
  std::cout << "Packing" << std::endl;
  for (size_t c = 0; c < 2; c++) {
    const size_t complex_offset_host = c * pixels * samples;
    const size_t complex_offset_device_packed = c * pixels_padded * samples_padded / CHAR_BIT;
    // process chunks
    for (size_t pixel_start = 0; pixel_start < pixels; pixel_start += pixels_per_chunk) {
      size_t local_npixels = pixels_per_chunk;
      // correct npixels in last chunk
      if (pixel_start + local_npixels > pixels) {
        local_npixels = pixels - pixel_start;
        // ensure any padded region is set to zero
        d_a_chunk.zero(chunk_size);
      }
      // copy chunk to device, row-by-row to handle padding
      for (size_t pixel = 0; pixel < local_npixels; pixel++) {
        const size_t d_offset = pixel * samples_padded;
        const size_t offset = (pixel_start + pixel) * samples + complex_offset_host;
        const size_t bytes_to_transfer = samples;

        cu::DeviceMemory d_a_chunk_slice(d_a_chunk, d_offset, bytes_to_transfer);
        stream.memcpyHtoDAsync(d_a_chunk_slice, static_cast<char *>(a_matrix_host) + offset, bytes_to_transfer);
      }
      // get offset for this chunk in a_packed
      cu::DeviceMemory d_a_packed_chunk(d_a_matrix_packed,
                                        pixel_start * samples_padded / CHAR_BIT + complex_offset_device_packed,
                                        local_npixels * samples_padded / CHAR_BIT);
      // run packing kernel
      ccglib::packing::Packing packing(local_npixels * samples_padded, device, stream);
      packing.Run(d_a_chunk, d_a_packed_chunk, ccglib::packing::pack, ccglib::packing::complex_first);
    }
  }

  // conjugate
  std::cout << "Conjugate" << std::endl;
  dim3 threads(256);
  dim3 grid(ccglib::helper::ceildiv(bytes_a_matrix_packed / 2, threads.x));
  cu::DeviceMemory d_a_matrix_packed_imag(d_a_matrix_packed, bytes_a_matrix_packed / 2, bytes_a_matrix_packed / 2);
  conjugate_1bit<<<grid, threads, 0, stream>>>(
      reinterpret_cast<unsigned *>(static_cast<hipDeviceptr_t>(d_a_matrix_packed_imag)), bytes_a_matrix_packed / 2);

  // transpose
  std::cout << "Transpose" << std::endl;
  ccglib::transpose::Transpose transpose(1, pixels_padded, samples_padded, tile_sizes.x, tile_sizes.z, 1, device,
                                         stream);
  transpose.Run(d_a_matrix_packed, d_a_transposed);

  // copy output to host
  std::cout << "Copy to host" << std::endl;
  cu::HostMemory a_matrix_output(bytes_a_matrix_packed);
  stream.memcpyDtoHAsync(a_matrix_output, d_a_transposed, bytes_a_matrix_packed);
  stream.synchronize();

  // write to disk
  std::cout << "Write to disk" << std::endl;
  std::ofstream out(path_a_matrix_out, std::ios::binary | std::ios::out);
  if (!out) {
    throw std::runtime_error("Failed to open output file: " + path_a_matrix_out);
  }
  out.write(static_cast<char *>(a_matrix_output), bytes_a_matrix_packed);
}
